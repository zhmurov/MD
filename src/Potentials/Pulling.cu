#include "hip/hip_runtime.h"
#include "Pulling.cuh"

Pulling::Pulling(MDData* mdd, float3* h_base_r0, int base_freq, float vel, float3* h_n, float* h_ks, int dcd_freq){

	this->mdd = mdd;
	this->h_base_r0 = h_base_r0;
	this->base_freq = base_freq;
	this->vel = vel;
	this->h_n = h_n;
	this->h_ks = h_ks;
	this->dcd_freq = dcd_freq;

	hipMalloc((void**)&d_base_r0, mdd->N*sizeof(float3));
	hipMemcpy(d_base_r0, h_base_r0, mdd->N*sizeof(float3), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_n, mdd->N*sizeof(float3));
	hipMemcpy(d_n, h_n, mdd->N*sizeof(float3), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_ks, mdd->N*sizeof(float));
	hipMemcpy(d_ks, h_ks, mdd->N*sizeof(float), hipMemcpyHostToDevice);

	this->base_displacement = base_displacement;
	base_displacement = 0.0f;

	this->aver_fmod = aver_fmod;
	aver_fmod = 0.0f;		// average fmod

	this->blockCount = (mdd->N-1)/DEFAULT_BLOCK_SIZE + 1;
	this->blockSize = DEFAULT_BLOCK_SIZE;

	FILE* data = fopen("force_extension.out", "w");
	fclose(data);

//FORCE
	h_fmod = (float*)calloc(mdd->N, sizeof(float));
	hipMalloc((void**)&d_fmod, mdd->N*sizeof(float));
	hipMemcpy(d_fmod, h_fmod, mdd->N*sizeof(float), hipMemcpyHostToDevice);

//ENERGY
	h_energy = (float*)calloc(mdd->N, sizeof(float));
	hipMalloc((void**)&d_energy, mdd->N*sizeof(float));
	hipMemcpy(d_energy, h_energy, mdd->N*sizeof(float), hipMemcpyHostToDevice);
}

Pulling::~Pulling(){
	free(h_energy);
	hipFree(d_energy);
}


__global__ void Pulling_kernel(float3* d_base_r0, float base_displacement, float3* d_n, float* d_ks, float* d_fmod){

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i < c_mdd.N){

		float4 ri = c_mdd.d_coord[i];
		float4 f = c_mdd.d_force[i];
		float3 r0 = d_base_r0[i];

		float ks = d_ks[i];
		float3 n = d_n[i];

		float4 dr;

		dr.x = r0.x + n.x*base_displacement - ri.x;
		dr.y = r0.y + n.y*base_displacement - ri.y;
		dr.z = r0.z + n.z*base_displacement - ri.z;

		f.x += ks*dr.x;
		f.y += ks*dr.y;
		f.z += ks*dr.z;

		c_mdd.d_force[i] = f;

		dr.w = sqrtf(dr.x*dr.x + dr.y*dr.y + dr.z*dr.z);
		d_fmod[i] = ks*dr.w;
	}
}

void Pulling::compute(){

	if(mdd->step % base_freq == 0){
		base_displacement = vel*mdd->dt*mdd->step;
	}
	if(mdd->step % dcd_freq == 0){
		printf("base_displacement = %f\n", base_displacement);
	}

	Pulling_kernel<<<this->blockCount, this->blockSize>>>(d_base_r0, base_displacement, d_n, d_ks, d_fmod);

	hipMemcpy(h_fmod, d_fmod, mdd->N*sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i < mdd->N; i++){
		aver_fmod += h_fmod[i];
	}

	// OUTPUT
	if(mdd->step % dcd_freq == 0){

		float dx, dy, dz, dr;
		hipMemcpy(mdd->h_coord, mdd->d_coord, mdd->N*sizeof(float4), hipMemcpyDeviceToHost);

		for(int i = 0; i < mdd->N; i++){
			if(h_ks[i] > 0.0f){
				dx = mdd->h_coord[i].x - h_base_r0[i].x;
				dy = mdd->h_coord[i].y - h_base_r0[i].y;
				dz = mdd->h_coord[i].z - h_base_r0[i].z;
				dr = sqrtf(dx*dx + dy*dy + dz*dz);
			}
		}
		aver_fmod /= float(dcd_freq);

		FILE* data = fopen("force_extension.out", "a");
		fprintf(data, "%12d\t", mdd->step);

		for(int i = 0; i < mdd->N; i++){
			if(h_ks[i] > 0.0f){
				fprintf(data, "%4.6f\t", base_displacement);
				fprintf(data, "%4.6f\t", aver_fmod);
				fprintf(data, "%4.6f", dr);
			}
		}
		fprintf(data, "\n");
		fclose(data);

		aver_fmod = 0.0f;
	}
}

/*
void PullingEnergykernel(){

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i < c_mdd.N){
		;
	}
}
*/

float Pulling::getEnergies(int energyId, int timestep){

	//PullingEnergykernel<<<this->blockCount, this->blockSize>>>();

	hipMemcpy(h_energy, d_energy, mdd->N*sizeof(float), hipMemcpyDeviceToHost);
	float energy_sum = 0.0f;

	for(int i = 0; i < mdd->N; i++){
		energy_sum += h_energy[i];
	}
	return energy_sum;
}

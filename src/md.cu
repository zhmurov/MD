#include "hip/hip_runtime.h"
/*
 * md.cu
 *
 *  Created on: 15.08.2012
 *      Author: zhmurov
 *  Changes: 16.08.2016
 *	Author: kir_min
 */

#include "md.cuh"

// Util
#include "Util/ReductionAlgorithms.cu"
#include "Util/ReductionAlgorithmsFloat4.cu"

// Potentials
#include "Potentials/BondsClass2Atom.cu"
//#include "Potentials/BondsClass2Pair.cu"
#include "Potentials/AngleClass2.cu"
#include "Potentials/GaussExcluded.cu"
#include "Potentials/Langevin.cu"
#include "Potentials/PPPM.cu"
#include "Potentials/Coulomb.cu"
#include "Potentials/FENE.cu"
#include "Potentials/LJP.cu"
#include "Potentials/Repulsive.cu"
#include "Potentials/PushingSphere.cu"
#include "Potentials/Indentation.cu"
#include "Potentials/Pulling.cu"

// Updaters
#include "Updaters/CoordinatesOutputDCD.cu"
#include "Updaters/EnergyOutput.cu"
#include "Updaters/PairlistUpdater.cu"
#include "Updaters/PairListL1.cu"
#include "Updaters/PairListL2.cu"
#include "Updaters/FixMomentum.cu"

// Integrators
#include "Integrators/LeapFrog.cu"
#include "Integrators/VelocityVerlet.cu"
#include "Integrators/LeapFrogNoseHoover.cu"
#include "Integrators/LeapFrogLangevin.cu"
#include "Integrators/LeapFrog_new.cu"
#include "Integrators/LeapFrogOverdamped.cu"
#include "Integrators/SteepestDescent.cu"

void dumpPSF(char* filename, TOPData &top){
	printf("Creating psf...\n");
	PSF psf;
	psf.natom = top.atomCount;
	psf.ntheta = 0;
	psf.nphi = 0;
	psf.nimphi = 0;
	psf.nnb = 0;
	psf.ncmap = 0;
	psf.atoms = (PSFAtom*)calloc(psf.natom, sizeof(PSFAtom));

	for(int i = 0; i < top.atomCount; i++){
		psf.atoms[i].id = top.atoms[i].id;
		psf.atoms[i].m = top.atoms[i].mass;

		sprintf(psf.atoms[i].name, "C");
		sprintf(psf.atoms[i].type, "%s", top.atoms[i].type);
		psf.atoms[i].q = top.atoms[i].charge;
		sprintf(psf.atoms[i].resName, "%s", top.atoms[i].resName);
		psf.atoms[i].resid = top.atoms[i].resid;
		sprintf(psf.atoms[i].segment, "%s", top.atoms[i].type);
	}

	psf.nbond = 0;

	int func_fene, func_bc2a;

	func_fene = getIntegerParameter(PARAMETER_FUNCTIONTYPE_FENE, DEFAULT_FUNCTIONTYPE_FENE);
	func_bc2a = getIntegerParameter(PARAMETER_FUNCTIONTYPE_BONDSCLASS2ATOM, DEFAULT_FUNCTIONTYPE_BONDSCLASS2ATOM);

	for(int i = 0; i < top.bondCount; i++){
		if ((top.bonds[i].func == func_fene) || (top.bonds[i].c0 == 1 && top.bonds[i].func == func_bc2a)){
			psf.nbond ++;
		}
	}
	psf.bonds = (PSFBond*)calloc(psf.nbond, sizeof(PSFBond));
	int currentBond = 0;

	for(int i = 0; i < top.bondCount; i++){
		if ((top.bonds[i].func == func_fene) || (top.bonds[i].c0 == 1 && top.bonds[i].func == func_bc2a)){
			psf.bonds[currentBond].i = getIndexInTOP(top.bonds[i].i, &top) + 1;
			psf.bonds[currentBond].j = getIndexInTOP(top.bonds[i].j, &top) + 1;
			currentBond++;
		}
	}

	writePSF(filename, &psf);
	free(psf.atoms);
	free(psf.bonds);
}

void readCoordinatesFromFile(char* filename, MDData mdd){
	XYZ xyz;
	readXYZ(filename, &xyz);
	for(int i = 0; i < xyz.atomCount; i++){
		mdd.h_coord[i].x = xyz.atoms[i].x/10.0;		// [angstr] -> [nm]
		mdd.h_coord[i].y = xyz.atoms[i].y/10.0;		// [angstr] -> [nm]
		mdd.h_coord[i].z = xyz.atoms[i].z/10.0;		// [angstr] -> [nm]		
	}
}

void checkCUDAError(const char* msg) {
	//hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error == hipSuccess)
		error = hipDeviceSynchronize();
	if (error == hipSuccess)
		error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CudaError: %s: %s\n", msg, hipGetErrorString(error));
		exit(0);
	}
}

void MDGPU::init()
{
	int i, j, b, p;

	initTimer();

	PDB pdbref;
	TOPData top;
	PARAMData par;

	char filename[FILENAME_LENGTH];
	getMaskedParameter(filename, PARAMETER_TOPOLOGY_FILENAME);
	readTOP(filename, &top);

	if(getYesNoParameter(PARAMETER_POTENTIAL_BONDSCLASS2ATOM, DEFAULT_POTENTIAL_BONDSCLASS2ATOM) || getYesNoParameter(PARAMETER_POTENTIAL_ANGLECLASS2, DEFAULT_POTENTIAL_ANGLECLASS2) || getYesNoParameter(PARAMETER_POTENTIAL_GAUSSEXCLUDED, DEFAULT_POTENTIAL_GAUSSEXCLUDED)){
		getMaskedParameter(filename, PARAMETER_PARAMETERS_FILENAME);
		readPARAM(filename, &par);
	}

	getMaskedParameter(filename, PARAMETER_PSF_OUTPUT_FILENAME);
	dumpPSF(filename, top);

	//TODO
	int feneFunc, ljFunc, repFunc; //protein
	int func_bc2a, func_ac2; //dna

	hipSetDevice(getIntegerParameter(PARAMETER_GPU_DEVICE));
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	mdd.N = top.atomCount;
	printf("mdd.N\t%d\n", mdd.N);
	mdd.widthTot = ((mdd.N-1)/DEFAULT_DATA_ALLIGN + 1)*DEFAULT_DATA_ALLIGN;
	mdd.dt = getFloatParameter(PARAMETER_TIMESTEP);
	mdd.numsteps = getIntegerParameter(PARAMETER_NUMSTEPS);

	mdd.ftm2v = FTM2V;

	mdd.bc.rlo.x = getFloatParameter("pbc_xlo", 0, 0);
	mdd.bc.rlo.y = getFloatParameter("pbc_ylo", 0, 0);
	mdd.bc.rlo.z = getFloatParameter("pbc_zlo", 0, 0);

	mdd.bc.rhi.x = getFloatParameter("pbc_xhi", 0, 0);
	mdd.bc.rhi.y = getFloatParameter("pbc_yhi", 0, 0);
	mdd.bc.rhi.z = getFloatParameter("pbc_zhi", 0, 0);

	mdd.bc.len.x = mdd.bc.rhi.x - mdd.bc.rlo.x;
	mdd.bc.len.y = mdd.bc.rhi.y - mdd.bc.rlo.y;
	mdd.bc.len.z = mdd.bc.rhi.z - mdd.bc.rlo.z;

	mdd.h_coord = (float4*)calloc(mdd.N, sizeof(float4));
	mdd.h_vel = (float4*)calloc(mdd.N, sizeof(float4));
	mdd.h_force = (float4*)calloc(mdd.N, sizeof(float4));
	mdd.h_mass = (float*)calloc(mdd.N, sizeof(float));
	mdd.h_charge = (float*)calloc(mdd.N, sizeof(float));
	mdd.h_atomTypes = (int*)calloc(mdd.N, sizeof(int));
	mdd.h_boxids = (int4*)calloc(mdd.N, sizeof(int4));

	getMaskedParameter(filename, PARAMETER_COORDINATES_FILENAME, "NONE");

	if(strncmp(filename, "NONE", 4) != 0){
		readCoordinatesFromFile(filename, mdd);
	}

	hipMalloc((void**)&mdd.d_coord, mdd.N*sizeof(float4));
	hipMalloc((void**)&mdd.d_vel, mdd.N*sizeof(float4));
	hipMalloc((void**)&mdd.d_force, mdd.N*sizeof(float4));
	hipMalloc((void**)&mdd.d_mass, mdd.N*sizeof(float));
	hipMalloc((void**)&mdd.d_charge, mdd.N*sizeof(float));
	hipMalloc((void**)&mdd.d_atomTypes, mdd.N*sizeof(int));
	hipMalloc((void**)&mdd.d_boxids, mdd.N*sizeof(int4));

	for(i = 0; i < mdd.N; i++){
		mdd.h_charge[i] = top.atoms[i].charge;
		mdd.h_atomTypes[i] = atoi(top.atoms[i].type) - 1; //TODO
	}

	for(i = 0; i < mdd.N; i++){
		mdd.h_mass[i] = top.atoms[i].mass;
	}

	float totalMass = 0.0f;
	for(i = 0; i < mdd.N; i++){
		totalMass += mdd.h_mass[i];
	}
	mdd.M = totalMass;

	int rseed = -getLongIntegerParameter(PARAMETER_RSEED);
	generateVelocities(getFloatParameter(PARAMETER_TEMPERATURE), &rseed);

	hipMemcpy(mdd.d_coord, mdd.h_coord, mdd.N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_force, mdd.h_force, mdd.N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_vel, mdd.h_vel, mdd.N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_atomTypes, mdd.h_atomTypes, mdd.N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_mass, mdd.h_mass, mdd.N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_charge, mdd.h_charge, mdd.N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_boxids, mdd.h_boxids, mdd.N*sizeof(int4), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(c_mdd), &mdd, sizeof(MDData), 0, hipMemcpyHostToDevice);

	int* fixedAtomsMask;
	fixedAtomsMask = (int*)calloc(mdd.N, sizeof(int));
	for(i = 0; i < mdd.N; i++){
		fixedAtomsMask[i] = 0;
	}

	if(getYesNoParameter(PARAMETER_FIX, DEFAULT_FIX) || getYesNoParameter(PARAMETER_PULLING, DEFAULT_PULLING)){

		getMaskedParameter(filename, PARAMETER_PDB_REFERENCE_FILENAME);
		readPDB(filename, &pdbref);

		if(getYesNoParameter(PARAMETER_FIX, DEFAULT_FIX)){
			for(i = 0; i < mdd.N; i++){
				fixedAtomsMask[i] = int(pdbref.atoms[i].beta);
			}
		}
	}

	char integ_str[PARAMETER_MAX_LENGTH];
	getMaskedParameter(integ_str, PARAMETER_INTEGRATOR);

	if(strcmp(integ_str, VALUE_INTEGRATOR_LEAP_FROG) == 0){
		integrator = new LeapFrog(&mdd, fixedAtomsMask);
	}else if (strcmp(integ_str, VALUE_INTEGRATOR_LEAP_FROG_NEW) == 0){
		int seed = getIntegerParameter(PARAMETER_RSEED);
		float temperature = getFloatParameter(PARAMETER_TEMPERATURE);
		integrator = new LeapFrog_new(&mdd, temperature, seed, fixedAtomsMask);
	}else if (strcmp(integ_str, VALUE_INTEGRATOR_LEAP_FROG_LANGEVIN) == 0){
		int seed = getIntegerParameter(PARAMETER_RSEED);
		float temperature = getFloatParameter(PARAMETER_TEMPERATURE);
		float damping = getFloatParameter(PARAMETER_LEAP_FROG_LANGEVIN_DAMPING, -1.0);
		integrator = new LeapFrogLangevin(&mdd, temperature, seed, fixedAtomsMask, damping);
	}else if (strcmp(integ_str, VALUE_INTEGRATOR_LEAP_FROG_OVERDUMPED) == 0){
		int seed = getIntegerParameter(PARAMETER_RSEED);
		float temperature = getFloatParameter(PARAMETER_TEMPERATURE);
		integrator = new LeapFrogOverdamped(&mdd, temperature, seed, fixedAtomsMask);
	}else if (strcmp(integ_str, VALUE_INTEGRATOR_VELOCITY_VERLET) == 0){
		integrator = new VelocityVerlet(&mdd, fixedAtomsMask);
	}else if (strcmp(integ_str, VALUE_INTEGRATOR_LEAP_FROG_NOSE_HOOVER) == 0){
		float tau = getFloatParameter(PARAMETER_NOSE_HOOVER_TAU);
		float T0 = getFloatParameter(PARAMETER_NOSE_HOOVER_T0);
		integrator = new LeapFrogNoseHoover(&mdd, tau, T0, fixedAtomsMask);
	}else if (strcmp(integ_str, VALUE_INTEGRATOR_STEEPEST_DESCENT) == 0){
		int seed = getIntegerParameter(PARAMETER_RSEED);
		float temperature = getFloatParameter(PARAMETER_TEMPERATURE);
		float maxForce = getFloatParameter(PARAMETER_STEEPEST_DESCENT_MAXFORCE);
		integrator = new SteepestDescent(&mdd, temperature, seed, maxForce, fixedAtomsMask);
	}else{
		DIE("Integrator was set incorrectly!\n");
	}

	if(getYesNoParameter(PARAMETER_LANGEVIN, DEFAULT_LANGEVIN)){
		float damping = getFloatParameter(PARAMETER_DAMPING);
		int seed = getIntegerParameter(PARAMETER_LANGEVIN_SEED);
		float temperature = getFloatParameter(PARAMETER_TEMPERATURE);
		potentials.push_back(new Langevin(&mdd, damping, seed, temperature));
	}

//=====================================================================
//DNA POTENTIALS
//=====================================================================

	//BondsClass2Atom potential
	if(getYesNoParameter(PARAMETER_POTENTIAL_BONDSCLASS2ATOM, DEFAULT_POTENTIAL_BONDSCLASS2ATOM)){

		func_bc2a = getIntegerParameter(PARAMETER_FUNCTIONTYPE_BONDSCLASS2ATOM, DEFAULT_FUNCTIONTYPE_BONDSCLASS2ATOM);

		int bondCountsPar = par.bondCount;
		int bondCountsTop = 0;

		for(b = 0; b < top.bondCount; b++){
			if(top.bonds[b].func == func_bc2a){
				bondCountsTop++;
			}
		}

		int4* pair;
		pair = (int4*)calloc(bondCountsTop, sizeof(int4));

		bondCountsTop = 0;
		for(i = 0; i < top.bondCount; i++){
			if(top.bonds[i].func == func_bc2a){
				pair[bondCountsTop].x = getIndexInTOP(top.bonds[i].i, &top);
				pair[bondCountsTop].y = getIndexInTOP(top.bonds[i].j, &top);
				pair[bondCountsTop].z = (int)top.bonds[i].c0 - 1;
				bondCountsTop++;
			}
		}

		float4* bondCoeffs;
		bondCoeffs = (float4*)calloc(bondCountsPar, sizeof(float4));

		for(i = 0; i < bondCountsPar; i++){
			bondCoeffs[i].x = par.bondCoeff[i].l0/10.0;		// [angstr] -> [nm]
			bondCoeffs[i].y = par.bondCoeff[i].k2*4.184*100.0;	// [kcal/(mol*angstr^2)] -> [kJ/(mol*nm^2)]
			bondCoeffs[i].z = par.bondCoeff[i].k3*4.184*1000.0;	// [kcal/(mol*angstr^3)] -> [kJ/(mol*nm^3)]
			bondCoeffs[i].w = par.bondCoeff[i].k4*4.184*10000.0;	// [kcal/(mol*angstr^4)] -> [kJ/(mol*nm^4)]
		}
		checkCUDAError("CUDA ERROR: before BondClass2Atom potential\n");
		potentials.push_back(new BondsClass2Atom(&mdd, bondCountsPar, bondCountsTop, pair, bondCoeffs));
		checkCUDAError("CUDA ERROR: after BondClass2Atom potential\n");
	}

	//AngleClass2 potential
	if(getYesNoParameter(PARAMETER_POTENTIAL_ANGLECLASS2, DEFAULT_POTENTIAL_ANGLECLASS2)){

		func_ac2 = getIntegerParameter(PARAMETER_FUNCTIONTYPE_ANGLECLASS2, DEFAULT_FUNCTIONTYPE_ANGLECLASS2);

		int angleCountsPar = par.angleCount;
		int angleCountsTop = 0;

		for(i = 0; i < top.angleCount; i++){
			if(top.angles[i].func == func_ac2){
				angleCountsTop++;
			}
		}

		int4* angle;
		angle = (int4*)calloc(angleCountsTop, sizeof(int4));

		angleCountsTop = 0;
		for(i = 0; i < top.angleCount; i++){
			if(top.angles[i].func == func_ac2){
				angle[i].x = getIndexInTOP(top.angles[i].i, &top);
				angle[i].y = getIndexInTOP(top.angles[i].j, &top);
				angle[i].z = getIndexInTOP(top.angles[i].k, &top);
				angle[i].w = (int)top.angles[i].c0 - 1;
				angleCountsTop++;
			}
		}

		float4* angleCoeffs;
		angleCoeffs = (float4*)calloc(angleCountsPar, sizeof(float4));

		for(i = 0; i < angleCountsPar; i++){
			angleCoeffs[i].x = par.angleCoeff[i].theta0;	// [degree]
			angleCoeffs[i].y = par.angleCoeff[i].k2*4.184;	// [kcal/(mol*rad^2)] -> [kJ/(mol*rad^2)]
			angleCoeffs[i].z = par.angleCoeff[i].k3*4.184;	// [kcal/(mol*rad^3)] -> [kJ/(mol*rad^3)]
			angleCoeffs[i].w = par.angleCoeff[i].k4*4.184;	// [kcal/(mol*rad^4)] -> [kJ/(mol*rad^4)]
		}

		checkCUDAError("CUDA ERROR: before AngleClass2 potential\n");
		potentials.push_back(new AngleClass2(&mdd, angleCountsPar, angleCountsTop, angle, angleCoeffs));
		checkCUDAError("CUDA ERROR: after AngleClass2 potential\n");
	}

	float dielectric = getFloatParameter(PARAMETER_DIELECTRIC, DEFAULT_DIELECTRIC);
	float coulCutoff = getFloatParameter(PARAMETER_COULOMB_CUTOFF);

	//Initialization of pairLists
	if(getYesNoParameter(PARAMETER_POTENTIAL_GAUSSEXCLUDED, DEFAULT_POTENTIAL_GAUSSEXCLUDED) || getYesNoParameter(PARAMETER_POTENTIAL_COULOMB, DEFAULT_POTENTIAL_COULOMB)){

		float gausExclCutoff = getFloatParameter(PARAMETER_NONBONDED_CUTOFF);
		float pairsCutoff = getFloatParameter(PARAMETER_PAIRLIST_CUTOFF);
		float possiblePairsCutoff = getFloatParameter(PARAMETER_POSSIBLE_PAIRLIST_CUTOFF);
		int possiblePairsFreq = getIntegerParameter(PARAMETER_POSSIBLE_PAIRLIST_FREQUENCE);
		int pairsFreq = getIntegerParameter(PARAMETER_PAIRLIST_FREQUENCE);

		std::vector<int2> exclusions(top.exclusionCount);
		for (i = 0; i < top.exclusionCount; i++){
			if(getIndexInTOP(top.exclusions[i].i, &top) < getIndexInTOP(top.exclusions[i].j, &top)){
				exclusions[i].x = getIndexInTOP(top.exclusions[i].i, &top);
				exclusions[i].y = getIndexInTOP(top.exclusions[i].j, &top);
			} else {
				exclusions[i].x = getIndexInTOP(top.exclusions[i].j, &top);
				exclusions[i].y = getIndexInTOP(top.exclusions[i].i, &top);
			}
		}
		std::sort(exclusions.begin(), exclusions.end(), &int2_comparatorEx);

		PairListL1* plistL1 = new PairListL1(&mdd, exclusions, possiblePairsCutoff, pairsCutoff, possiblePairsFreq);
		PairListL2* plistL2 = new PairListL2(&mdd, plistL1->d_pairs, pairsCutoff, coulCutoff, pairsFreq);
		updaters.push_back(plistL1);
		updaters.push_back(plistL2);

		//GaussExcluded potential
		if(getYesNoParameter(PARAMETER_POTENTIAL_GAUSSEXCLUDED, DEFAULT_POTENTIAL_GAUSSEXCLUDED)){

			int typeCount = 1;
			bool boo;
			for (i = 1; i < top.atomCount; i++){
				for(j = 0; j < i; j++){
					if (atoi(top.atoms[j].type) == atoi(top.atoms[i].type)){ 
						boo = false;
						break;
					}else{
						boo = true;
					}
				}
				if (boo) {
					typeCount++;
				}
			}

			for (i = 1; i < top.atomCount; i++){
				if(typeCount < atoi(top.atoms[i].type)){
					typeCount = atoi(top.atoms[i].type);
				}
			}
			printf("typeCount = %d\n", typeCount);
	
			GaussExCoeff* gaussExCoeff;
			gaussExCoeff = (GaussExCoeff*)calloc(typeCount*typeCount, sizeof(GaussExCoeff));

			for(i = 0; i < typeCount; i++){
				for(j = 0; j < typeCount; j++){
					for(int k = 0; k < par.ljCount; k++){
						if((i == par.lj_RepulsiveCoeff[k].i - 1 && j == par.lj_RepulsiveCoeff[k].j - 1) || (j == par.lj_RepulsiveCoeff[k].i - 1 && i == par.lj_RepulsiveCoeff[k].j - 1)){
							gaussExCoeff[i+j*typeCount].l = par.lj_RepulsiveCoeff[k].l;
							gaussExCoeff[i+j*typeCount].A = par.lj_RepulsiveCoeff[k].A*4.184/pow(10.0, gaussExCoeff[i+j*typeCount].l);	// [kcal/mol*angstr^l] -> [kJ/mol*nm^l];
						}
					}
					for(int k = 0; k < par.gaussCount; k++){
						if((i == par.gaussCoeff[k].i - 1 && j == par.gaussCoeff[k].j - 1) || (j == par.gaussCoeff[k].i - 1 && i == par.gaussCoeff[k].j - 1)){
							gaussExCoeff[i+j*typeCount].numberGaussians = par.gaussCoeff[k].numberGaussians;
							gaussExCoeff[i+j*typeCount].B = (float*)calloc(par.gaussCoeff[k].numberGaussians, sizeof(float));
							gaussExCoeff[i+j*typeCount].C = (float*)calloc(par.gaussCoeff[k].numberGaussians, sizeof(float));
							gaussExCoeff[i+j*typeCount].R = (float*)calloc(par.gaussCoeff[k].numberGaussians, sizeof(float));
							for(int l = 0; l < par.gaussCoeff[k].numberGaussians; l++){
								gaussExCoeff[i+j*typeCount].B[l] = par.gaussCoeff[k].B[l]*4.184;		// [kcal/mol] -> [kJ/mol]
								gaussExCoeff[i+j*typeCount].C[l] = par.gaussCoeff[k].C[l]*100.0;		// [1/angstr^2] -> [1/nm^2]
								gaussExCoeff[i+j*typeCount].R[l] = par.gaussCoeff[k].R[l]/10.0;			// [angstr] -> [nm]
							}
						}		
					}
				}
			}
	
			float cutoff = getFloatParameter(PARAMETER_NONBONDED_CUTOFF);

			checkCUDAError("CUDA ERROR: before GaussExcluded potential\n");
			potentials.push_back(new GaussExcluded(&mdd, cutoff, typeCount, gaussExCoeff, plistL2));
			checkCUDAError("CUDA ERROR: after GaussExcluded potential\n");
		}

		//Coulomb potential
		//PPPM potential
		if(getYesNoParameter(PARAMETER_POTENTIAL_COULOMB, DEFAULT_POTENTIAL_COULOMB) && getYesNoParameter(PARAMETER_POTENTIAL_PPPM, DEFAULT_POTENTIAL_PPPM)){

			PPPM* pppm = new PPPM(&mdd, dielectric, coulCutoff);

			checkCUDAError("CUDA ERROR: before PPPM potential\n");
			potentials.push_back(pppm);
			checkCUDAError("CUDA ERROR: after PPPM potential\n");

			checkCUDAError("CUDA ERROR: before Coulomb potential\n");
			potentials.push_back(new Coulomb(&mdd, plistL2, pppm->get_alpha(), dielectric, coulCutoff));
			checkCUDAError("CUDA ERROR: after Coulomb potential\n");
		}
	}


//=====================================================================
//PROTEIN POTENTIALS
//=====================================================================

	//FENE potential
	if(getYesNoParameter(PARAMETER_POTENTIAL_FENE, DEFAULT_POTENTIAL_FENE)){

		feneFunc = getIntegerParameter(PARAMETER_FUNCTIONTYPE_FENE, DEFAULT_FUNCTIONTYPE_FENE);
		float feneKs = getFloatParameter(PARAMETER_KS_FENE);	//spring constant					[kJ/(mol*nm^2)]
		float feneR = getFloatParameter(PARAMETER_R_FENE);	//tolerance to the change of the covalent bond length	[nm]

		int feneCount = 0;
		for(b = 0; b < top.bondCount; b++){
			if(top.bonds[b].func == feneFunc){
				feneCount++;
			}
		}

		int2* feneBonds;
		feneBonds = (int2*)calloc(feneCount, sizeof(int2));

		float* feneBondsR0;
		feneBondsR0 = (float*)calloc(feneCount, sizeof(float));

		feneCount = 0;
		for(b = 0; b < top.bondCount; b++){
			if(top.bonds[b].func == feneFunc){
				feneBonds[feneCount].x = getIndexInTOP(top.bonds[b].i, &top);
				feneBonds[feneCount].y = getIndexInTOP(top.bonds[b].j, &top);
				feneBondsR0[feneCount] = top.bonds[b].c0/10.0f; 			// [angstr]->[nm]
				feneCount++;
			}
		}

		checkCUDAError("CUDA ERROR: before FENE potential\n");
		potentials.push_back(new FENE(&mdd, feneKs, feneR, feneCount, feneBonds, feneBondsR0));
		checkCUDAError("CUDA ERROR: after FENE potential\n");
	}

	//LennardJones potential
	if(getYesNoParameter(PARAMETER_POTENTIAL_LENNARDJONES, DEFAULT_POTENTIAL_LENNARDJONES)){

		ljFunc = getIntegerParameter(PARAMETER_FUNCTIONTYPE_LENNARDJONES, DEFAULT_FUNCTIONTYPE_LENNARDJONES);

		int ljCount = 0;
		for(p = 0; p < top.pairsCount; p++){
			if(top.pairs[p].func == ljFunc){
				ljCount++;
			}
		}

		int2* ljPairs;
		ljPairs = (int2*)calloc(ljCount, sizeof(int2));

		float* ljPairsR0;
		ljPairsR0 = (float*)calloc(ljCount, sizeof(float));		// equilibrium distance

		float* ljPairsEps;
		ljPairsEps = (float*)calloc(ljCount, sizeof(float));		// epsilon

		ljCount = 0;
		for(p = 0; p < top.pairsCount; p++){
			if(top.pairs[p].func == ljFunc){
				ljPairs[ljCount].x = getIndexInTOP(top.pairs[p].i, &top);
				ljPairs[ljCount].y = getIndexInTOP(top.pairs[p].j, &top);
				ljPairsR0[ljCount] = top.pairs[p].c0/10.0; 	// [angstr]->[nm]
				ljPairsEps[ljCount] = top.pairs[p].c1*4.184; 	// [kCal/mol]->[kJ/mol]
				ljCount++;
			}
		}
		checkCUDAError("CUDA ERROR: before LennardJones potential\n");
		potentials.push_back(new LJP(&mdd, ljCount, ljPairs, ljPairsR0, ljPairsEps));
		checkCUDAError("CUDA ERROR: after LennardJones potential\n");
	}

	//Repulsive potential (PPPM and Coulumb have to be off)
	if(getYesNoParameter(PARAMETER_POTENTIAL_REPULSIVE, DEFAULT_POTENTIAL_REPULSIVE) && !getYesNoParameter(PARAMETER_POTENTIAL_GAUSSEXCLUDED, DEFAULT_POTENTIAL_GAUSSEXCLUDED) && !getYesNoParameter(PARAMETER_POTENTIAL_COULOMB, DEFAULT_POTENTIAL_COULOMB)){

		repFunc = getIntegerParameter(PARAMETER_FUNCTIONTYPE_REPULSIVE, DEFAULT_FUNCTIONTYPE_REPULSIVE);

		std::vector<int2> exclusions(top.exclusionCount);

		for(i = 0; i < top.exclusionCount; i++){
			if(top.exclusions[i].func == repFunc){
				if(getIndexInTOP(top.exclusions[i].i, &top) < getIndexInTOP(top.exclusions[i].j, &top)){
					exclusions[i].x = getIndexInTOP(top.exclusions[i].i, &top);
					exclusions[i].y = getIndexInTOP(top.exclusions[i].j, &top);
				}else{
					exclusions[i].x = getIndexInTOP(top.exclusions[i].j, &top);
					exclusions[i].y = getIndexInTOP(top.exclusions[i].i, &top);
				}
			}
		}

		std::sort(exclusions.begin(), exclusions.end(), &int2_comparatorEx);

		//pairList1
		float possiblePairsCutoff = getFloatParameter(PARAMETER_POSSIBLE_PAIRLIST_CUTOFF);
		int possiblePairsFreq = getIntegerParameter(PARAMETER_POSSIBLE_PAIRLIST_FREQUENCE);
		//pairList2
		float pairsCutoff = getFloatParameter(PARAMETER_PAIRLIST_CUTOFF);
		int pairsFreq = getIntegerParameter(PARAMETER_PAIRLIST_FREQUENCE);

		float nbCutoff = getFloatParameter(PARAMETER_NONBONDED_CUTOFF);

		PairListL1* plistL1 = new PairListL1(&mdd, exclusions, possiblePairsCutoff, pairsCutoff, possiblePairsFreq);
		PairListL2* plistL2 = new PairListL2(&mdd, plistL1->d_pairs, pairsCutoff, nbCutoff, pairsFreq);
		updaters.push_back(plistL1);
		updaters.push_back(plistL2);

		float repEps = getFloatParameter(PARAMETER_REPULSIVE_EPSILON);
		float repSigm = getFloatParameter(PARAMETER_REPULSIVE_SIGMA);

		checkCUDAError("CUDA ERROR: before Repulsive potential\n");
		potentials.push_back(new Repulsive(&mdd, plistL2, nbCutoff, repEps, repSigm));
		checkCUDAError("CUDA ERROR: after Repulsive potential\n");
	}

//=====================================================================
//OTHER POTENTIALS (NO DNA, NO PROTEIN)
//=====================================================================

	//PushingSphere potential
	if(getYesNoParameter(PARAMETER_PUSHING_SPHERE, DEFAULT_PUSHING_SPHERE)){

		float psR0 = getFloatParameter(PARAMETER_PUSHING_SPHERE_RADIUS0);

		float psR = getFloatParameter(PARAMETER_PUSHING_SPHERE_RADIUS, -1.0);
		float psV = getFloatParameter(PARAMETER_PUSHING_SPHERE_SPEED, 0.0);

		if((psR > 0) && (psV == 0)){
			psV = (psR0 - psR)/mdd.numsteps;
		}

		float4 pscenterPoint;
		getVectorParameter(PARAMETER_PUSHING_SPHERE_CENTER_POINT, &pscenterPoint.x, &pscenterPoint.y, &pscenterPoint.z);
		float psUpdate = getIntegerParameter(PARAMETER_PUSHING_SPHERE_UPDATE_FREQ);
		float psSigma = getFloatParameter(PARAMETER_PUSHING_SPHERE_SIGMA);
		float psEpsilon = getFloatParameter(PARAMETER_PUSHING_SPHERE_EPSILON);
		int lj_or_harmonic = 0;
		if(getYesNoParameter(PARAMETER_PUSHING_SPHERE_HARMONIC, DEFAULT_PUSHING_SPHERE_HARMONIC)){
			lj_or_harmonic = 1; 		
		}
		char psFilename[1024];
		getMaskedParameter(psFilename, PARAMETER_PUSHING_SPHERE_OUTPUT_FILENAME);
		
		int* push_mask;
		push_mask = (int*)calloc(top.atomCount, sizeof(int));

		if(getYesNoParameter(PARAMETER_PUSHING_SPHERE_MASK, DEFAULT_PUSHING_SPHERE_MASK)){
			char psPDBFilename[1024];
			getMaskedParameter(psPDBFilename, PARAMETER_PUSHING_SPHERE_MASK_PDB_FILENAME);
			PDB push_maskPDB;
			readPDB(psPDBFilename, &push_maskPDB);
			for(i = 0; i < push_maskPDB.atomCount; i++){
				if((int)push_maskPDB.atoms[i].occupancy == 1){
					push_mask[i] = 1;	
				}			
			}
		}else{
			for(i = 0; i < top.atomCount; i++){
				if(atoi(top.atoms[i].type) == 1){
					push_mask[i] = 1;
				}			
			}		
		}

		checkCUDAError("CUDA ERROR: before PushingSphere potential\n");
		potentials.push_back(new PushingSphere(&mdd, psR0, psV, pscenterPoint, psUpdate, psSigma, psEpsilon, psFilename, lj_or_harmonic, push_mask));
		checkCUDAError("CUDA ERROR: after PushingSphere potential\n");
	}

	//Pulling potential
	if(getYesNoParameter(PARAMETER_PULLING, DEFAULT_PULLING)){

		getMaskedParameter(filename, PARAMETER_PDB_REFERENCE_FILENAME);
		if(mdd.N != pdbref.atomCount){
			printf("Error: number of atoms in top is not equal the number of atoms in pdbref\n");
		}

		float3* pullBaseR0;
		pullBaseR0 = (float3*)calloc(pdbref.atomCount, sizeof(float3));
		int pullBaseFreq = getIntegerParameter(PARAMETER_PULLING_BASE_DISPLACEMENT_FREQUENCY);
		float3* pullN;
		pullN = (float3*)calloc(pdbref.atomCount, sizeof(float3));
		float pullVel = getFloatParameter(PARAMETER_PULLING_VELOCITY);
		float* pullKs;
		pullKs = (float*)calloc(pdbref.atomCount, sizeof(float));
		int dcdFreq = getIntegerParameter(PARAMETER_DCD_OUTPUT_FREQUENCY);

		//pdbref.atoms.occupancy - spring constant
		//pdbref.atoms.x(y,z) - force vector

		for(i = 0; i < pdbref.atomCount; i++){
			if(pdbref.atoms[i].occupancy != 0.0f){
				pullBaseR0[i].x = mdd.h_coord[i].x;
				pullBaseR0[i].y = mdd.h_coord[i].y;
				pullBaseR0[i].z = mdd.h_coord[i].z;

				pullN[i].x = pdbref.atoms[i].x;
				pullN[i].y = pdbref.atoms[i].y;
				pullN[i].z = pdbref.atoms[i].z;

				pullKs[i] = pdbref.atoms[i].occupancy;
			}
		}

		checkCUDAError("CUDA ERROR: before Pulling potential\n");
		potentials.push_back(new Pulling(&mdd, pullBaseR0, pullBaseFreq, pullVel, pullN, pullKs, dcdFreq));
		checkCUDAError("CUDA ERROR: after Pulling potential\n");
	}

	//Indentation potential
	if(getYesNoParameter(PARAMETER_INDENTATION, DEFAULT_INDENTATION)){

		int atomCount = 0;
		for (i = 0; i < top.atomCount; i++){
			if (strcmp(top.atoms[i].type, "4") == 0){
				atomCount++;
			}
		}
		printf("Indentation atomCount = %d\n", atomCount);

		float ind_tip_radius = getFloatParameter(PARAMETER_INDENTATION_TIP_RADIUS);
		float3 ind_tip_coord;
		getVectorParameter(PARAMETER_INDENTATION_TIP_COORD, &ind_tip_coord.x, &ind_tip_coord.y, &ind_tip_coord.z);
		float3 ind_base_coord;
		getVectorParameter(PARAMETER_INDENTATION_BASE_COORD, &ind_base_coord.x, &ind_base_coord.y, &ind_base_coord.z);
		int ind_base_freq = getIntegerParameter(PARAMETER_INDENTATION_BASE_DISPLACEMENT_FREQUENCY);
		float3 ind_n;
		getVectorParameter(PARAMETER_INDENTATION_N, &ind_n.x, &ind_n.y, &ind_n.z);
		float ind_vel = getFloatParameter(PARAMETER_INDENTATION_VELOCITY);

		ind_vel = (float(ind_base_freq)/60.86)*ind_vel;
		printf("vel = %f\n", ind_vel);	

		float ind_ks = getFloatParameter(PARAMETER_INDENTATION_KSPRING);
		float ind_eps = getFloatParameter(PARAMETER_INDENTATION_EPSILON);
		float ind_sigm = getFloatParameter(PARAMETER_INDENTATION_SIGMA);

		//surface
		float3 sf_coord;
		getVectorParameter(PARAMETER_SURFACE_COORD, &sf_coord.x, &sf_coord.y, &sf_coord.z);
		float3 sf_n;
		getVectorParameter(PARAMETER_SURFACE_N, &sf_n.x, &sf_n.y, &sf_n.z);
		float sf_eps = getFloatParameter(PARAMETER_SURFACE_EPSILON);
		float sf_sigm = getFloatParameter(PARAMETER_SURFACE_SIGMA);


		int dcd_freq = getIntegerParameter(PARAMETER_DCD_OUTPUT_FREQUENCY);
		char pdb_cant_filename[FILENAME_LENGTH];
		getMaskedParameter(pdb_cant_filename, PARAMETER_PDB_CANTILEVER_OUTPUT_FILENAME);
		char dcd_cant_filename[FILENAME_LENGTH];
		getMaskedParameter(dcd_cant_filename, PARAMETER_DCD_CANTILEVER_OUTPUT_FILENAME);

		//TODO TODO TODO
		//cantilever
		PDB pdb_cant;
		readPDB(pdb_cant_filename, &pdb_cant);

/*
		int atomCount_cant = 2;

		pdb_cant.atomCount = atomCount_cant;
		pdb_cant.atoms = (PDBAtom*)calloc(pdb_cant.atomCount, sizeof(PDBAtom));
		//tip
		pdb_cant.atoms[0].id = 1;
		strcpy(pdb_cant.atoms[0].name, "TIP");
		pdb_cant.atoms[0].chain = 'T';
		strcpy(pdb_cant.atoms[0].resName, "tip");
		pdb_cant.atoms[0].altLoc = ' ';
		pdb_cant.atoms[0].resid = 0;
		pdb_cant.atoms[0].x = ind_tip_coord.x*10.0;	// [nm] -> [angstr]
		pdb_cant.atoms[0].y = ind_tip_coord.y*10.0;	// [nm] -> [angstr]
		pdb_cant.atoms[0].z = ind_tip_coord.z*10.0;	// [nm] -> [angstr]
		//base
		pdb_cant.atoms[1].id = 2;
		strcpy(pdb_cant.atoms[1].name, "BASE");
		pdb_cant.atoms[1].chain = 'B';
		strcpy(pdb_cant.atoms[1].resName, "bas");
		pdb_cant.atoms[1].altLoc = ' ';
		pdb_cant.atoms[1].resid = 0;
		pdb_cant.atoms[1].x = ind_base_coord.x*10.0;	// [nm] -> [angstr]
		pdb_cant.atoms[1].y = ind_base_coord.y*10.0;	// [nm] -> [angstr]
		pdb_cant.atoms[1].z = ind_base_coord.z*10.0;	// [nm] -> [angstr]

		writePDB(pdb_cant_filename, &pdb_cant);
*/

		checkCUDAError("CUDA ERROR: before Indentation potential\n");
		potentials.push_back(new Indentation(&mdd, atomCount, ind_tip_radius, ind_tip_coord, ind_base_coord, ind_base_freq, ind_n, ind_vel, ind_ks, ind_eps, ind_sigm, sf_coord, sf_n, sf_eps, sf_sigm, dcd_freq, dcd_cant_filename));
		checkCUDAError("CUDA ERROR: after Indentation potential\n");
	}


//UPDATERS
	updaters.push_back(new CoordinatesOutputDCD(&mdd));
	updaters.push_back(new EnergyOutput(&mdd, &potentials));
	
	if(getYesNoParameter(PARAMETER_FIX_MOMENTUM, DEFAULT_FIX_MOMENTUM)){
		updaters.push_back(new FixMomentum(&mdd, getIntegerParameter(PARAMETER_FIX_MOMENTUM_FREQUENCE)));
	}

	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);
	printf("\nMEMORY USED: %f%%\n", 100.0f*(1.0f - float(free_mem)/float(total_mem)));
}

void MDGPU::generateVelocities(float T, int * rseed){
	printf("Generating velocities at temperature T=%fK.\n", T);
	int i;
	if(T < 0){
		DIE("Negative value for temperature is set (T = %fK).", T);
	} else
	if(T == 0){
		for(i = 0; i < mdd.N; i++){
			mdd.h_vel[i].x = 0.0;
			mdd.h_vel[i].y = 0.0;
			mdd.h_vel[i].z = 0.0;
		}
	} else {
		for(i = 0; i < mdd.N; i++){
			double var = sqrt(BOLTZMANN_CONSTANT*T/mdd.h_mass[i]);
			mdd.h_vel[i].x = var*gasdev(rseed);
			mdd.h_vel[i].y = var*gasdev(rseed);
			mdd.h_vel[i].z = var*gasdev(rseed);
		}
	}
	float Temp = 0.0f;
	float Vav = 0.0f;
	int freq = getIntegerParameter(PARAMETER_ENERGY_OUTPUT_FREQUENCY);
	FILE* file = fopen("vels.dat", "w");
	for(i = 0; i < mdd.N; i++){
		mdd.h_vel[i].w = mdd.h_vel[i].x*mdd.h_vel[i].x + mdd.h_vel[i].y*mdd.h_vel[i].y + mdd.h_vel[i].z*mdd.h_vel[i].z;
		Vav += sqrtf(mdd.h_vel[i].w);
		fprintf(file, "%f\n", sqrtf(mdd.h_vel[i].w));
		Temp += mdd.h_vel[i].w*mdd.h_mass[i];;
		mdd.h_vel[i].w *= freq;
	}
	fclose(file);
	Temp /= mdd.N;
	Temp /= 3.0*BOLTZMANN_CONSTANT;
	Vav /= mdd.N;
	printf("Temperature of the system: %f (average velocity %f)\n", Temp, Vav);
}

void MDGPU::compute()
{
	mdd.step = 0;
	printTime(mdd.step);
	int numsteps = mdd.numsteps;
	int nav = numsteps;
	int i;
	int u;		// updater
	int p;		// potential

	// nav - lowest updaters frequency
	for(u = 0; u != updaters.size(); u++){
		if(nav > updaters[u]->getFrequence()){
			nav = updaters[u]->getFrequence();
		}
	}
	// TODO
	for(p = 0; p != potentials.size(); p++){
		potentials[p]->compute();
	}
	while(mdd.step <= numsteps){
		for(u = 0; u != updaters.size(); u++){
			if(mdd.step % updaters[u]->getFrequence() == 0){
				updaters[u]->update();
			}
		}
		for(i = 0; i < nav; i++){
			integrator->integrateStepOne();
			for(p = 0; p != potentials.size(); p++){
				potentials[p]->compute();
				checkCUDAError("CUDA ERROR: after potential inside MDGPU:compute()\n");
			}
			integrator->integrateStepTwo();
			mdd.step++;
		}
	}
	
	//XYZ-File ending coord
	if(getYesNoParameter(PARAMETER_OUTPUT_XYZ, DEFAULT_OUTPUT_XYZ)){
		char filename[FILENAME_LENGTH];
		getMaskedParameter(filename, PARAMETER_OUTPUT_XYZ_FILENAME);
		FILE * file;
		file = fopen(filename, "w");
		fprintf(file, "%d\n", mdd.N);
		fprintf(file, "Created by mdd.cu\n");
		for(i = 0; i < mdd.N; i++){
			fprintf(file, "%s\t%f\t%f\t%f\n", "P", mdd.h_coord[i].x*10.0, mdd.h_coord[i].y*10.0, mdd.h_coord[i].z*10.0);
		}
		fclose(file);
	}
}

MDGPU::~MDGPU()
{

	free(mdd.h_coord);
	free(mdd.h_vel);
	free(mdd.h_force);
	free(mdd.h_mass);
	free(mdd.h_charge);
	free(mdd.h_atomTypes);

	hipFree(mdd.d_coord);
	hipFree(mdd.d_vel);
	hipFree(mdd.d_force);
	hipFree(mdd.d_mass);
	hipFree(mdd.d_charge);
	hipFree(mdd.d_atomTypes);
}

void compute(){

	MDGPU mdgpu;
	mdgpu.init();
	mdgpu.compute();
	hipDeviceReset();
}


#include "hip/hip_runtime.h"
/*
 * md.cu
 *
 *  Created on: 15.08.2012
 *      Author: zhmurov
 */

#include "md.cuh"

// Util
#include "Util/ReductionAlgorithms.cu"
#include "Util/ReductionAlgorithmsFloat4.cu"

// Potentials
#include "Potentials/BondsClass2Atom.cu"
#include "Potentials/BondsClass2Pair.cu"
#include "Potentials/AngleClass2.cu"
#include "Potentials/GaussExcluded.cu"
#include "Potentials/Langevin.cu"
#include "Potentials/PPPM.cu"
#include "Potentials/Coulomb.cu"

// Updaters
#include "Updaters/CoordinatesOutputDCD.cu"
#include "Updaters/EnergyOutput.cu"
#include "Updaters/PairlistUpdater.cu"
#include "Updaters/PairListL1.cu"
#include "Updaters/PairListL2.cu"
#include "Updaters/FixMomentum.cu"

// Integrators
#include "Integrators/LeapFrog.cu"
#include "Integrators/VelocityVerlet.cu"
#include "Integrators/LeapFrogNoseHoover.cu"

void dumpPSF(char* filename, ReadTopology &top){
	PSF psf;
	psf.natoms = top.atomCount;
	psf.ntheta = 0;
	psf.nphi = 0;
	psf.nimphi = 0;
	psf.nnb = 0;
	psf.ncmap = 0;
	psf.atoms = (PSFAtom*)calloc(psf.natom, sizeof(PSFAtom));

	int i, j;
	for(i = 0; i < top.atomCount; i++){
		psf.atoms[i].id = top.atoms[i].id;
		psf.atoms[i].m = top.masses[j].mass;

		sprintf(psf.atoms[i].name, "C");
		sprintf(psf.atoms[i].type, "%d", top.atoms[i].type);
		psf.atoms[i].q = top.atoms[i].charge;
		if(top.atoms[i].resid == 0){
			sprintf(psf.atoms[i].resName, "ION");
		} else {
			sprintf(psf.atoms[i].resName, "DNA");
		}
		psf.atoms[i].resid = top.atoms[i].resid;
		sprintf(psf.atoms[i].segment, "%d", top.atoms[i].id);
	}

	psf.bondCount = 0;
	for(i = 0; i < top.bondCount; i++){
		if(top.bonds[i].func == 1){
			psf.nbond ++;
		}
	}
	psf.bonds = (PSFBond*)calloc(psf.nbond, sizeof(PSFBond));
	int currentBond = 0;
	for(i = 0; i < top.bondCount; i++){
		if(top.bonds[i].func == 1){
			psf.bonds[currentBond].i = top.bonds[i].i;
			psf.bonds[currentBond].j = top.bonds[i].j;
			currentBond++;
		}
	}

	writePSF(filename, &psf);
	free(psf.atoms);
	free(psf.bonds);
}

void readCoordinatesFromFile(char* filename){
	XYZ xyz;
	readXYZ(filename, &xyz);
	int i;
	for(i = 0; i < xyz.atomCount; i++){
		mdd.h_coord[i].x = xyz.atoms[i].x;
		mdd.h_coord[i].y = xyz.atoms[i].y;
		mdd.h_coord[i].z = xyz.atoms[i].z;
	}
}



void MDGPU::init()
{
	parseParametersFile(argv[1], argc, argv);

	TOPData top;
	PARAMData par;

	char filename[FILENAME_LENGTH];
	getMaskedParameter(filename, PARAMETER_TOPOLOGY_FILENAME);
	readTOP(filename, &top);

	getMaskedParameter(filename, PARAMETER_PARAMETERS_FILENAME);
	ReadParameters par(filename, &par);

	getMaskedParameter(filename, PARAMETER_PSF_OUTPUT_FILENAME);
	dumpPSF(filename, top);

	hipSetDevice(getIntegerParameter(PARAMETER_GPU_DEVICE));
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	mdd.N = top.atomCount;
	mdd.widthTot = ((mdd.N-1)/DEFAULT_DATA_ALLIGN + 1)*DEFAULT_DATA_ALLIGN;
	mdd.dt = getFloatParameter(PARAMETER_TIMESTEP);
	mdd.numsteps = getIntegerParameter(PARAMETER_NUMSTEPS);

	mdd.ftm2v = FTM2V;

	mdd.bc.rlo.x = getFloatParameter("pbc_xlo", 0, 0);
	mdd.bc.rlo.y = getFloatParameter("pbc_ylo", 0, 0);
	mdd.bc.rlo.z = getFloatParameter("pbc_zlo", 0, 0);

	mdd.bc.rhi.x = getFloatParameter("pbc_xhi", 0, 0);
	mdd.bc.rhi.y = getFloatParameter("pbc_yhi", 0, 0);
	mdd.bc.rhi.z = getFloatParameter("pbc_zhi", 0, 0);

	mdd.bc.len.x = mdd.bc.rhi.x - mdd.bc.rlo.x;
	mdd.bc.len.y = mdd.bc.rhi.y - mdd.bc.rlo.y;
	mdd.bc.len.z = mdd.bc.rhi.z - mdd.bc.rlo.z;

	mdd.h_coord = (float4*)calloc(mdd.N, sizeof(float4));
	mdd.h_vel = (float4*)calloc(mdd.N, sizeof(float4));
	mdd.h_force = (float4*)calloc(mdd.N, sizeof(float4));
	mdd.h_mass = (float*)calloc(mdd.N, sizeof(float));
	mdd.h_charge = (float*)calloc(mdd.N, sizeof(float));
	mdd.h_atomTypes = (int*)calloc(mdd.N, sizeof(int));
	mdd.h_boxids = (int4*)calloc(mdd.N, sizeof(int4));

	getMaskedParameter(filename, PARAMETER_COORDINATES_FILENAME, "NONE");

	if(strncmp(filename, "NONE", 4) != 0){
		readCoordinatesFromFile(filename);
	}

	hipMalloc((void**)&mdd.d_coord, mdd.N*sizeof(float4));
	hipMalloc((void**)&mdd.d_vel, mdd.N*sizeof(float4));
	hipMalloc((void**)&mdd.d_force, mdd.N*sizeof(float4));
	hipMalloc((void**)&mdd.d_mass, mdd.N*sizeof(float));
	hipMalloc((void**)&mdd.d_charge, mdd.N*sizeof(float));
	hipMalloc((void**)&mdd.d_atomTypes, mdd.N*sizeof(int));
	hipMalloc((void**)&mdd.d_boxids, mdd.N*sizeof(int4));


	int i, j;
	for(i = 0; i < mdd.N; i++){
		mdd.h_charge[i] = top.atoms[i].charge;
		mdd.h_atomTypes[i] = top.atoms[i].type - 1;
	}

	for(i = 0; i < mdd.N; i++){
		for(j = 0; j < top.atomCount; j++){
			if(top.atoms[i].type == top.atoms[j].type){
				mdd.h_mass[i] = top.atoms[j].mass;
			}
		}
	}
	double totalMass = 0.0;
	for(i = 0; i < mdd.N; i++){
		totalMass += mdd.h_mass[i];
	}
	mdd.M = totalMass;
	

	int rseed = -getLongIntegerParameter(PARAMETER_RSEED);
	generateVelocities(getFloatParameter(PARAMETER_TEMPERATURE), &rseed);

	hipMemcpy(mdd.d_coord, mdd.h_coord, mdd.N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_force, mdd.h_force, mdd.N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_vel, mdd.h_vel, mdd.N*sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_atomTypes, mdd.h_atomTypes, mdd.N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_mass, mdd.h_mass, mdd.N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_charge, mdd.h_charge, mdd.N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mdd.d_boxids, mdd.h_boxids, mdd.N*sizeof(int4), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(c_mdd), &mdd, sizeof(MDData), 0, hipMemcpyHostToDevice);
	hipBindTexture(0, t_coord, mdd.d_coord, mdd.N*sizeof(float4));
	hipBindTexture(0, t_charges, mdd.d_charge, mdd.N*sizeof(float));
	hipBindTexture(0, t_atomTypes, mdd.d_atomTypes, mdd.N*sizeof(int));

	char integ_str[PARAMETER_MAX_LENGTH];
	getMaskedParameter(integ_str, PARAMETER_INTEGRATOR);
	if (strcmp(integ_str, VALUE_INTEGRATOR_LEAP_FROG) == 0) {
		integrator = new LeapFrog(&mdd);
	} else if (strcmp(integ_str, VALUE_INTEGRATOR_VELOCITY_VERLET) == 0) {
		integrator = new VelocityVerlet(&mdd);
	} else if (strcmp(integ_str, VALUE_INTEGRATOR_LEAP_FROG_NOSE_HOOVER) == 0) {
		integrator = new LeapFrogNoseHoover(&mdd);
	} else {
		DIE("Integrator was set incorrectly!");
	}

	if(getYesNoParameter(PARAMETER_LANGEVIN, DEFAULT_LANGEVIN)){
		float damping = getFloatParameter(PARAMETER_DAMPING);
		int seed = getIntegerParameter(PARAMETER_LANGEVIN_SEED);
		float temperature = getFloatParameter(PARAMETER_TEMPERATURE);
		potentials.push_back(new Langevin(&mdd, damping, seed, temperature));
	}


/*


	// Computational arrays
	ComputationalArrays ca(&top, &par);

	std::vector<int3> bonds;
	std::vector<Coeffs> bond_coeffs;
	std::vector<int2> exclusions;

	// Add potentials, updaters and integrators

	

	ca.GetBondList(string(BOND_CLASS2_STRING), &bonds, &bond_coeffs);
//	potentials.push_back(new BondsClass2Atom(&mdd, bonds, bond_coeffs));
	potentials.push_back(new BondsClass2Pair(&mdd, bonds, bond_coeffs));

	potentials.push_back(new AngleClass2(&mdd, top, par));
	

	// Init pair lists

	float gausExclCutoff = getFloatParameter(PARAMETER_NONBONDED_CUTOFF);
	float coulCutoff = getFloatParameter(PARAMETER_COULOMB_CUTOFF);
	float pairsCutoff = getFloatParameter(PARAMETER_PAIRLIST_CUTOFF);
	float possiblePairsCutoff = getFloatParameter(PARAMETER_POSSIBLE_PAIRLIST_CUTOFF);

	int possiblePairsFreq = getIntegerParameter(PARAMETER_POSSIBLE_PAIRLIST_FREQUENCE);
	int pairsFreq = getIntegerParameter(PARAMETER_PAIRLIST_FREQUENCE);

	std::vector<int> exclTypes;
	if(hasParameter(PARAMETER_EXCLUDE_BOND_TYPES)) {
		exclTypes = getIntegerArrayParameter(PARAMETER_EXCLUDE_BOND_TYPES);
	}
	ca.GetExclusionList(&exclusions, &exclTypes);

	PairListL1* plistL1 = new PairListL1(&mdd, exclusions, possiblePairsCutoff, pairsCutoff, possiblePairsFreq);
	PairListL2* plistL2 = new PairListL2(&mdd, plistL1->d_pairs, pairsCutoff, coulCutoff, pairsFreq);
	updaters.push_back(plistL1);
	updaters.push_back(plistL2);
	if(coulCutoff - gausExclCutoff > 10.0f){
		PairListL2* plistGausExcl = new PairListL2(&mdd, plistL2->d_pairs, coulCutoff, gausExclCutoff, pairsFreq);
		potentials.push_back(new GaussExcluded(&mdd, top, par, plistGausExcl));
		updaters.push_back(plistGausExcl);
	} else {
		potentials.push_back(new GaussExcluded(&mdd, top, par, plistL2));
	}

	float dielectric = getFloatParameter(PARAMETER_DIELECTRIC, DEFAULT_DIELECTRIC);
	PPPM* pppm = new PPPM(&mdd, dielectric, coulCutoff);
	potentials.push_back(pppm);
	potentials.push_back(new Coulomb(&mdd, plistL2, pppm->get_alpha(), dielectric, coulCutoff));


	updaters.push_back(new CoordinatesOutputDCD(&mdd));
	updaters.push_back(new EnergyOutput(&mdd, &potentials));
	
	if(getYesNoParameter(PARAMETER_FIX_MOMENTUM, DEFAULT_FIX_MOMENTUM)){
		updaters.push_back(new FixMomentum(&mdd, getIntegerParameter(PARAMETER_FIX_MOMENTUM_FREQUENCE)));
	}*/
}

void MDGPU::generateVelocities(float T, int * rseed){
	printf("Generating velocities at temperature T=%fK.\n", T);
	int i;
	if(T < 0){
		DIE("Negative value for temperature is set (T = %fK).", T);
	} else
	if(T == 0){
		for(i = 0; i < mdd.N; i++){
			mdd.h_vel[i].x = 0.0;
			mdd.h_vel[i].y = 0.0;
			mdd.h_vel[i].z = 0.0;
		}
	} else {
		for(i = 0; i < mdd.N; i++){
			double var = sqrt(BOLTZMANN_CONSTANT*T/mdd.h_mass[i]);
			mdd.h_vel[i].x = var*gasdev(rseed);
			mdd.h_vel[i].y = var*gasdev(rseed);
			mdd.h_vel[i].z = var*gasdev(rseed);
		}
	}
	float Temp = 0.0f;
	float Vav = 0.0f;
	int freq = getIntegerParameter(PARAMETER_ENERGY_OUTPUT_FREQUENCY);
	FILE* file = fopen("vels.dat", "w");
	for(i = 0; i < mdd.N; i++){
		mdd.h_vel[i].w = mdd.h_vel[i].x*mdd.h_vel[i].x + mdd.h_vel[i].y*mdd.h_vel[i].y + mdd.h_vel[i].z*mdd.h_vel[i].z;
		Vav += sqrtf(mdd.h_vel[i].w);
		fprintf(file, "%f\n", sqrtf(mdd.h_vel[i].w));
		Temp += mdd.h_vel[i].w*mdd.h_mass[i];;
		mdd.h_vel[i].w *= freq;
	}
	fclose(file);
	Temp /= mdd.N;
	Temp /= 3.0*BOLTZMANN_CONSTANT;
	Vav /= mdd.N;
	printf("Temperature of the system: %f (average velocity %f)\n", Temp, Vav);
}


void checkCUDAError(const char* msg) {
	//hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error == hipSuccess)
		error = hipDeviceSynchronize();
	if (error == hipSuccess)
		error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CudaError: %s: %s\n", msg, hipGetErrorString(error));
exit(0);
	}
}

void MDGPU::compute()
{
	mdd.step = 0;
	int numsteps = mdd.numsteps;
	int nav = numsteps;
	int i;
	int u, p;
	for(u = 0; u != updaters.size(); u++){
		if(nav > updaters[u]->getFrequence()){
			nav = updaters[u]->getFrequence();
			//hipDeviceSynchronize();
		}
	}

	//hipDeviceSynchronize();
	for(p = 0; p != potentials.size(); p++){
		potentials[p]->compute(&mdd);
//checkCUDAError("Here!");
//printf("bla!");
		//hipDeviceSynchronize();
	}

	/*hipMemcpy(mdd.h_force, mdd.d_force, mdd.N*sizeof(float4), hipMemcpyDeviceToHost);
	FILE* file = fopen("forces.dat", "w");
	for(i = 0; i < mdd.N; i++){
		fprintf(file, "%f %f %f\n", mdd.h_force[i].x, mdd.h_force[i].y, mdd.h_force[i].z);
	}
	fclose(file);
	file = fopen("coords.dat", "w");
	for(i = 0; i < mdd.N; i++){
		fprintf(file, "%f %f %f\n", mdd.h_coord[i].x, mdd.h_coord[i].y, mdd.h_coord[i].z);
	}
	fclose(file);
	exit(0);*/

	for(mdd.step = 0; mdd.step <= numsteps; mdd.step += nav){
		for(u = 0; u != updaters.size(); u++){
			if(mdd.step % updaters[u]->getFrequence() == 0){
				updaters[u]->update(&mdd);
				//hipDeviceSynchronize();
			}
		}
		for(i = 0; i < nav; i++){
			integrator->integrate_step_one(&mdd);
			//hipDeviceSynchronize();
			for(p = 0; p != potentials.size(); p++){
				potentials[p]->compute(&mdd);
				//hipDeviceSynchronize();
			}

			integrator->integrate_step_two(&mdd);
		}
	}

}

MDGPU::~MDGPU()
{
	free(mdd.h_coord);
	free(mdd.h_vel);
	free(mdd.h_force);
	free(mdd.h_mass);
	free(mdd.h_charge);
	free(mdd.h_atomTypes);

	hipFree(mdd.d_coord);
	hipFree(mdd.d_vel);
	hipFree(mdd.d_force);
	hipFree(mdd.d_mass);
	hipFree(mdd.d_charge);
	hipFree(mdd.d_atomTypes);
}

void compute(ReadTopology &top, ReadParameters &par){

	MDGPU mdgpu;
	mdgpu.init();
	mdgpu.compute();
	hipDeviceReset();
}


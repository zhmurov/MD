#include "hip/hip_runtime.h"
/*
 * EnergyOutput.cu
 *
 *  Created on: 21.08.2012
 *      Author: zhmurov
 */

#include "EnergyOutput.cuh"

EnergyOutput::EnergyOutput(MDData *mdd, std::vector<IPotential*>* potentials){
	this->mdd = mdd;
	this->frequence = getIntegerParameter(PARAMETER_ENERGY_OUTPUT_FREQUENCY);
	this->potentials = potentials;
	getMaskedParameter(filename, PARAMETER_ENERGY_OUTPUT_FILENAME);
	FILE* file = fopen(filename, "w");
	fclose(file);
}

EnergyOutput::~EnergyOutput(){
}

void EnergyOutput::update(){

	FILE* file = fopen(filename, "a");

	int p, i;
	double temp = 0.0f;		//temperature [K]

	hipMemcpy(mdd->h_vel, mdd->d_vel, mdd->N*sizeof(float4), hipMemcpyDeviceToHost);

	// TODO ENERGY_OUTPUT_VELOCITY_WARNING
	double velThresholdSq = ENERGY_OUTPUT_VELOCITY_WARNING*(float)this->frequence;
	velThresholdSq = velThresholdSq*velThresholdSq;

	for(i = 0; i < mdd->N; i++){
		temp += mdd->h_vel[i].w*mdd->h_mass[i];
		if(mdd->h_vel[i].w > velThresholdSq){
			printf("WARNING: Velocity of the particle %d is larger than %f nm/ps\n", i, ENERGY_OUTPUT_VELOCITY_WARNING);
		}
		mdd->h_vel[i].w = 0.0f;
	}
	hipMemcpy(mdd->d_vel, mdd->h_vel, mdd->N*sizeof(float4), hipMemcpyHostToDevice);
	temp /= ((float)mdd->N)*((float)this->frequence)*3.0f*BOLTZMANN_CONSTANT;

	printf("%*s%*s",
			ENERGY_OUTPUT_WIDTH, ENERGY_OUTPUT_STEP,
			ENERGY_OUTPUT_WIDTH, ENERGY_OUTPUT_TEMPERATURE);
	for(p = 0; p != (*potentials).size(); p++){
		for(i = 0; i < (*potentials)[p]->getEnergyCount(); i++){
			printf("%*s", ENERGY_OUTPUT_WIDTH, (*potentials)[p]->getEnergyName(i).c_str());
		}
	}
	printf("%*s\n", ENERGY_OUTPUT_WIDTH, ENERGY_OUTPUT_TOTAL);

	printf("%*d%*f",
			ENERGY_OUTPUT_WIDTH, mdd->step,
			ENERGY_OUTPUT_WIDTH, temp);
	fprintf(file, "%d\t%f\t", mdd->step, temp);
	float totalEnergy = 0;
	for(p = 0; p != (*potentials).size(); p++){
		for(i = 0; i < (*potentials)[p]->getEnergyCount(); i++){
			float energy = (*potentials)[p]->getEnergies(i, mdd->step);
			totalEnergy += energy;
			printf("%*f", ENERGY_OUTPUT_WIDTH, energy);
			fprintf(file, "%f\t", energy);
		}
	}
	printf("%*f\n", ENERGY_OUTPUT_WIDTH, totalEnergy);
	fprintf(file, "%f\n", totalEnergy);
	fclose(file);
	printTime(mdd->step);
	printEstimatedTimeleft((float)mdd->step/(float)mdd->numsteps);
}
